#include "hip/hip_runtime.h"
#include "DomainSignatureCollection.h"
#include "DomainCollection.h"
#include "Timer.h"
#include "args.h"
#include "amgx_c.h"
#include <chrono>
#include <cmath>
#include <iostream>
//#include <mpi.h>
#include <string>
#include <fstream>
#include <unistd.h>
#ifndef M_PIl
#define M_PIl 3.141592653589793238462643383279502884L /* pi */
#endif

// =========== //
// main driver //
// =========== //

using namespace std;
void print_callback(const char *msg, int length)
{
    cout << msg;
}
int main(int argc, char *argv[])
{
	using namespace std::chrono;


	MPI_Init(NULL, NULL);
	int num_procs;
	MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

	int my_global_rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &my_global_rank);

	// parse input
	args::ArgumentParser parser("");
	args::HelpFlag       help(parser, "help", "Display this help menu", {'h', "help"});

	args::ValueFlag<int> f_n(parser, "n", "number of cells in the x direction, in each domain",
	                         {'n'});
	args::ValueFlag<string> f_mesh(parser, "file_name", "read in a mesh", {"mesh"});
	args::ValueFlag<int>    f_square(parser, "num_domains",
	                              "create a num_domains x num_domains square of grids", {"square"});
	args::ValueFlag<int> f_amr(parser, "num_domains", "create a num_domains x num_domains square "
	                                                  "of grids, and a num_domains*2 x "
	                                                  "num_domains*2 refined square next to it",
	                           {"amr"});
	args::Flag f_outclaw(parser, "outclaw", "output amrclaw ascii file", {"outclaw"});
#ifdef HAVE_VTK
	args::Flag f_outvtk(parser, "", "output to vtk format", {"outvtk"});
#endif
	args::ValueFlag<int> f_l(parser, "n", "run the program n times and print out the average",
	                         {'l'});
	args::ValueFlag<string> f_m(parser, "matrix filename", "the file to write the matrix to",
	                            {'m'});
	args::ValueFlag<string> f_s(parser, "solution filename", "the file to write the solution to",
	                            {'s'});
	args::ValueFlag<string> f_resid(parser, "residual filename",
	                                "the file to write the residual to", {"residual"});
	args::ValueFlag<string> f_error(parser, "error filename", "the file to write the error to",
	                                {"error"});
	args::ValueFlag<string> f_r(parser, "rhs filename", "the file to write the rhs vector to",
	                            {'r'});
	args::ValueFlag<string> f_g(parser, "gamma filename", "the file to write the gamma vector to",
	                            {'g'});
	args::ValueFlag<string> f_read_gamma(parser, "gamma filename",
	                                     "the file to read gamma vector from", {"readgamma"});
	args::ValueFlag<string> f_flux(parser, "flux filename", "the file to write flux difference to",
	                               {"flux"});
	args::ValueFlag<string> f_p(parser, "preconditioner filename",
	                            "the file to write the preconditioner to", {'p'});
	args::ValueFlag<double> f_t(
	parser, "tolerance", "set the tolerance of the iterative solver (default is 1e-10)", {'t'});
	args::ValueFlag<double> f_omega(
	parser, "tolerance", "set the tolerance of the iterative solver (default is 1e-10)", {"omega"});
	args::ValueFlag<int> f_d(
	parser, "row", "pin gamma value to zero (by modifying that row of the schur compliment matrix)",
	{'z'});
	args::ValueFlag<int> f_div(parser, "divide", "use iterative method", {"divide"});
	args::Flag           f_wrapper(parser, "wrapper", "use a function wrapper", {"wrap"});
	args::Flag           f_blockcrs(parser, "wrapper", "use a function wrapper", {"blockcrs"});
	args::Flag           f_crs(parser, "wrapper", "use a function wrapper", {"crs"});
	args::Flag           f_gauss(parser, "gauss", "solve gaussian function", {"gauss"});
	args::Flag           f_zero(parser, "gauss", "solve gaussian function", {"zero"});
	args::Flag           f_prec(parser, "prec", "use block diagonal preconditioner", {"prec"});
	args::Flag           f_precblockj(parser, "prec", "use block diagonal jacobi preconditioner",
	                        {"precblockj"});
	args::Flag f_precj(parser, "prec", "use block diagonal jacobi preconditioner", {"precj"});
	args::Flag f_precmuelu(parser, "prec", "use AMG preconditioner", {"muelu"});
	args::Flag f_precddmg(parser, "prec", "use AMG preconditioner", {"ddmg"});
	args::Flag f_neumann(parser, "neumann", "use neumann boundary conditions", {"neumann"});
	args::Flag f_cg(parser, "gmres", "use CG for iterative solver", {"cg"});
	args::Flag f_gmres(parser, "gmres", "use GMRES for iterative solver", {"gmres"});
	args::Flag f_lsqr(parser, "gmres", "use GMRES for iterative solver", {"lsqr"});
	args::Flag f_rgmres(parser, "rgmres", "use GCRO-DR (Recycling GMRES) for iterative solver",
	                    {"rgmres"});
	args::Flag f_bicg(parser, "gmres", "use BiCGStab for iterative solver", {"bicg"});
	args::Flag f_zerou(parser, "zerou", "modify matrix so that the sum of the solution is zero",
	                   {"nozerou"});
	args::Flag f_nozerof(parser, "zerou", "don't  make the rhs match the boundary conditions",
	                     {"nozerof"});
	args::Flag f_pingamma(parser, "pingamma", "pin the first gamma to zero", {"pingamma"});
	args::Flag f_lu(parser, "lu", "use KLU solver", {"klu"});
	args::Flag f_mumps(parser, "lu", "use MUMPS solver", {"mumps"});
	args::Flag f_basker(parser, "lu", "use Basker solver", {"basker"});
	args::Flag f_superlu(parser, "lu", "use SUPERLU solver", {"superlu"});
	args::Flag f_ilu(parser, "ilu", "use incomplete LU preconditioner", {"ilu"});
	args::Flag f_riluk(parser, "ilu", "use RILUK preconditioner", {"riluk"});
	args::Flag f_iter(parser, "iterative", "use iterative method", {"iterative"});

	if (argc < 5) {
		if (my_global_rank == 0) std::cout << parser;
		return 0;
	}
	try {
		parser.ParseCLI(argc, argv);
	} catch (args::Help) {
		if (my_global_rank == 0) std::cout << parser;
		return 0;
	} catch (args::ParseError e) {
		if (my_global_rank == 0) {
			std::cerr << e.what() << std::endl;
			std::cerr << parser;
		}
		return 1;
	} catch (args::ValidationError e) {
		if (my_global_rank == 0) {
			std::cerr << e.what() << std::endl;
			std::cerr << parser;
		}
		return 1;
	}

	bool direct_solve = (f_lu || f_superlu || f_mumps || f_basker);
	bool use_crs = (f_crs || direct_solve || f_ilu || f_riluk || f_precj || f_precmuelu || f_prec);

	DomainSignatureCollection dsc;
	if (f_mesh) {
		string d = args::get(f_mesh);
		dsc      = DomainSignatureCollection(d, my_global_rank);
	} else if (f_amr) {
		int d = args::get(f_amr);
		dsc   = DomainSignatureCollection(d, d, my_global_rank, true);
	} else {
		int d = args::get(f_square);
		dsc   = DomainSignatureCollection(d, d, my_global_rank);
	}
	if (f_div) {
		for (int i = 0; i < args::get(f_div); i++) {
			dsc.divide();
		}
	}
	if (f_neumann) {
		dsc.setNeumann();
		if (!f_pingamma && !f_zerou) {
		dsc.setZeroPatch();
		}
	}
	// Set the number of discretization points in the x and y direction.
	int nx          = args::get(f_n);
	int ny          = args::get(f_n);
	int total_cells = dsc.num_global_domains * nx * ny;
	cerr << "Total cells: " << total_cells << endl;

	if (dsc.num_global_domains < num_procs) {
		std::cerr << "number of domains must be greater than or equal to the number of processes\n";
		return 1;
	}
	// partition domains
	if (num_procs > 1) {
		dsc.zoltanBalance();
	}

	double tol = 1e-10;
	if (f_t) {
		tol = args::get(f_t);
	}

	int loop_count = 1;
	if (f_l) {
		loop_count = args::get(f_l);
	}

	string save_matrix_file = "";
	if (f_m) {
		save_matrix_file = args::get(f_m);
	}

	string save_solution_file = "";
	if (f_s) {
		save_solution_file = args::get(f_s);
	}

	string save_residual_file = "";
	if (f_resid) {
		save_residual_file = args::get(f_resid);
	}
	string save_error_file = "";
	if (f_error) {
		save_error_file = args::get(f_error);
	}
	string save_rhs_file = "";
	if (f_r) {
		save_rhs_file = args::get(f_r);
	}
	string save_gamma_file = "";
	if (f_g) {
		save_gamma_file = args::get(f_g);
	}

	string save_prec_file = "";
	if (f_p) {
		save_prec_file = args::get(f_p);
	}

	// the functions that we are using
	function<double(double, double)> ffun;
	function<double(double, double)> gfun;
	function<double(double, double)> nfunx;
	function<double(double, double)> nfuny;

	if (f_zero) {
		ffun  = [](double x, double y) { return 0; };
		gfun  = [](double x, double y) { return 0; };
		nfunx = [](double x, double y) { return 0; };
		nfuny = [](double x, double y) { return 0; };
	} else if (f_gauss) {
		gfun
		= [](double x, double y) { return exp(cos(10 * M_PIl * x)) - exp(cos(11 * M_PIl * y)); };
		ffun = [](double x, double y) {
			return 100 * M_PIl * M_PIl * (pow(sin(10 * M_PIl * x), 2) - cos(10 * M_PIl * x))
			       * exp(cos(10 * M_PIl * x))
			       + 121 * M_PIl * M_PIl * (cos(11 * M_PIl * y) - pow(sin(11 * M_PIl * y), 2))
			         * exp(cos(11 * M_PIl * y));
		};
		nfunx = [](double x, double y) {
			return -10 * M_PIl * sin(10 * M_PIl * x) * exp(cos(10 * M_PIl * x));
		};

		nfuny = [](double x, double y) {
			return 11 * M_PIl * sin(11 * M_PIl * y) * exp(cos(11 * M_PIl * y));
		};
	} else {
		ffun = [](double x, double y) {
			return -5 * M_PIl * M_PIl * sinl(M_PIl * y) * cosl(2 * M_PIl * x);
		};
		gfun = [](double x, double y) { return sinl(M_PIl * y) * cosl(2 * M_PIl * x); };
		nfunx
		= [](double x, double y) { return -2 * M_PIl * sinl(M_PIl * y) * sinl(2 * M_PIl * x); };
		nfuny = [](double x, double y) { return M_PIl * cosl(M_PIl * y) * cosl(2 * M_PIl * x); };
	}

    //library handles
    AMGX_Mode mode;
    AMGX_config_handle cfg;
    AMGX_resources_handle rsrc;
    AMGX_matrix_handle gA;
    AMGX_vector_handle gb, gx;
    AMGX_solver_handle solver;
    mode = AMGX_mode_dDDI;
    //status handling
    AMGX_SOLVE_STATUS status;
    /* init */
    AMGX_SAFE_CALL(AMGX_initialize());
    AMGX_SAFE_CALL(AMGX_initialize_plugins());
    /* system */
    AMGX_SAFE_CALL(AMGX_register_print_callback(&print_callback));
    AMGX_SAFE_CALL(AMGX_install_signal_handler());
    /* create resources, matrix, vector and solver */
    AMGX_config_create_from_file(&cfg, "amgx.json");
    AMGX_resources_create_simple(&rsrc, cfg);
    AMGX_matrix_create(&gA, rsrc, mode);
    AMGX_vector_create(&gx, rsrc, mode);
    AMGX_vector_create(&gb, rsrc, mode);
    AMGX_solver_create(&solver, rsrc, mode, cfg);

	Tools::Timer timer;
	for (int loop = 0; loop < loop_count; loop++) {
		timer.start("Domain Initialization");

		DomainCollection dc(dsc, nx);
		if (f_neumann) {
			if (f_neumann && f_zerou) {
				dc.setZeroU();
			}
		}

		if (f_neumann) {
			dc.initNeumann(ffun, gfun, nfunx, nfuny, f_amr);
		} else {
			dc.initDirichlet(ffun, gfun);
			dc.amr = f_amr;
		}

		timer.stop("Domain Initialization");

		// Create a map that will be used in the iterative solver

        int num_rows = nx*dsc.iface_map_vec.size();
		// Create the gamma and diff vectors
		double*                   gamma = new double[num_rows];
            for(int i=0;i<num_rows;i++){
                gamma[i]=0;
            }
		double*                   r     = new double[num_rows];
		double*                   x     = new double[num_rows];
		double*                   d     = new double[num_rows];
		double*                   diff  = new double[num_rows];
		double*                   b     = new double[num_rows];
		AmgxCrs                   A;

		if (f_neumann && !f_nozerof) {
			double fdiff = (dc.integrateBoundaryFlux() - dc.integrateF()) / dc.area();
			if (my_global_rank == 0) cout << "Fdiff: " << fdiff << endl;
            dc.zeroF(fdiff);
		}
		timer.start("Complete Solve");
		if (dsc.num_global_domains != 1) {
			// do iterative solve

			// Get the b vector
			dc.solveWithInterface(gamma, b);

			///////////////////
			// setup start
			///////////////////
			timer.start("Linear System Setup");

            timer.start("Matrix Formation");

            A= dc.formCRSMatrix();

            timer.stop("Matrix Formation");
            timer.start("AMGX Setup");

            
            AMGX_SAFE_CALL(AMGX_vector_upload(gb,num_rows,1,(void*)b));
            AMGX_SAFE_CALL(AMGX_vector_set_zero(gx,num_rows,1));
            AMGX_SAFE_CALL(AMGX_matrix_upload_all(gA,num_rows,A.nnz,1,1,&A.row_ptrs[0],&A.cols[0],(void*)&A.data[0],nullptr));
            AMGX_SAFE_CALL(AMGX_solver_setup(solver, gA));

            timer.stop("AMGX Setup");

			timer.stop("Linear System Setup");

		    timer.start("Gamma Solve");
            AMGX_SAFE_CALL(AMGX_solver_solve_with_0_initial_guess(solver,gb,gx));
            AMGX_SAFE_CALL(AMGX_vector_download(gx,(void*)gamma));

		    timer.stop("Gamma Solve");
		}

		// Do one last solve
		timer.start("Patch Solve");

		dc.solveWithInterface(gamma, diff);

		timer.stop("Patch Solve");

		dc.residual();
		double ausum2 = dc.integrateAU();
		double fsum2  = dc.integrateF();
		double bflux  = dc.integrateBoundaryFlux();
		if (my_global_rank == 0) {
			std::cout << u8"Σf-Au: " << fsum2 - ausum2 << endl;
			std::cout << u8"Σf: " << fsum2 << endl;
			std::cout << u8"ΣAu: " << ausum2 << endl;
			if (f_neumann) {
				std::cout << u8"∮ du/dn: " << bflux << endl;
				std::cout << u8"∮ du/dn - Σf: " << bflux - fsum2 << endl;
				std::cout << u8"∮ du/dn - ΣAu: " << bflux - ausum2 << endl;
			}
		}
        /*
           TODO iterative
		if (f_iter && !direct_solve) {
			timer.start("Iterative Refinement Step");
			dc.residual();
			dc.swapResidSol();

			if (dsc.num_global_domains != 1) {
				x->putScalar(0);
				dc.solveWithInterface(*x, *r);
				// op->apply(*gamma, *r);
				// r->update(1.0, *b, -1.0);

				solver->reset(Belos::ResetType::Problem);
				if (f_wrapper) {
					((FuncWrap *) op.getRawPtr())->setB(r);
				}
				problem->setProblem(x, r);
				solver->setProblem(problem);
				solver->solve();
			}
			dc.solveWithInterface(*x, *d);
			dc.sumResidIntoSol();
			timer.stop("Iterative Refinement Step");
		}
        */

		///////////////////
		// solve end
		///////////////////
		timer.stop("Complete Solve");

		// Calcuate error
        double exact_norm;
        double diff_norm;

		if (f_neumann) {
			double uavg = dc.integrateU() / dc.area();
			double eavg = dc.integrateExact() / dc.area();

			if (my_global_rank == 0) {
				cout << "Average of computed solution: " << uavg << endl;
				cout << "Average of exact solution: " << eavg << endl;
			}

			exact_norm = dc.exactNorm(eavg);
			diff_norm  = dc.diffNorm(uavg, eavg);
		} else {
			exact_norm = dc.exactNorm();
			diff_norm  = dc.diffNorm();
		}

		double residual = dc.residual();
		double fnorm    = dc.fNorm();
		double ausum    = dc.integrateAU();
		double fsum     = dc.integrateF();
		if (my_global_rank == 0) {
			std::cout << std::scientific;
			std::cout.precision(13);
			std::cout << "Error: " << diff_norm / exact_norm << endl;
			std::cout << "Residual: " << residual / fnorm << endl;
			std::cout << u8"ΣAu-Σf: " << ausum - fsum << endl;
			// if (f_neumann) {
			//	std::cout << u8"∮ du/dn - ΣAu: " << dc.sumBoundaryFlux() - ausum << endl;
			//}
			cout.unsetf(std::ios_base::floatfield);
		}
		if (save_solution_file != "") {
			ofstream out_file(save_solution_file);
			dc.outputSolution(out_file);
			out_file.close();
			if (f_amr) {
				ofstream out_file(save_solution_file + ".amr");
				dc.outputSolutionRefined(out_file);
				out_file.close();
			}
		}
		if (save_residual_file != "") {
			ofstream out_file(save_residual_file);
			dc.outputResidual(out_file);
			out_file.close();
			if (f_amr) {
				ofstream out_file(save_residual_file + ".amr");
				dc.outputResidualRefined(out_file);
				out_file.close();
			}
		}
		if (save_error_file != "") {
			ofstream out_file(save_error_file);
			dc.outputError(out_file);
			out_file.close();
			if (f_amr) {
				ofstream out_file(save_error_file + ".amr");
				dc.outputErrorRefined(out_file);
				out_file.close();
			}
		}
#ifdef HAVE_VTK
		if (f_outvtk) {
			dc.outputVTK();
		}
#endif
		cout.unsetf(std::ios_base::floatfield);
	}

	if (my_global_rank == 0) {
		cout << timer;
	}
	return 0;
}
